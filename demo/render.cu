#include "hip/hip_runtime.h"
/*
 *  First Cuda Program for Path Tracing
 *  Author: Hong Zhang
 *  Date: 2022/12/12
 */


#include "utils/include/alice_cuda_image.h"
#include "core/include/alice_cuda_render.h"
#include "core/include/alice_cuda_camera.h"

#include "interface/include/window.h"
#include "interface/include/imgui_widgets.h"
#include "interface/include/texture_cuda_gl.h"

int main() {
    // create an empty image
    int img_w = 1200;
    int img_h = 800;
    int img_c = 3;
    glm::vec2 resolution{img_w, img_h};

    // 1. create Window
    ALICE_TRACER::Window window{};
    window.initWindow(img_w, img_h);
    ALICE_TRACER::ImGUIWidget widgets;
    widgets.initImGui();

    auto * res_img = new ALICE_TRACER::Image(img_w, img_h, img_c);
    ALICE_TRACER::Texture texture{res_img};

    // start the timer
    clock_t start, stop;
    start = clock();

    int tile_x = 16;
    int tile_y = 16;
    dim3 blocks(img_w/tile_x + 1, img_h/tile_y + 1);
    dim3 threads(tile_x, tile_y);

    // camera
    ALICE_TRACER::Camera * camera;
    hipMallocManaged(&camera, sizeof(ALICE_TRACER::Camera));
    camera->pos_ = glm::vec3(0.f, 0.f, 4.f);
    camera->look_at_ = glm::vec3(0.f);
    camera->head_up_ = glm::vec3(0.f, 1.f, 0.f);
    camera->resolution_ = resolution;
    camera->fov_ = glm::radians(60.f);


    ALICE_TRACER::render<<<blocks,threads>>>(res_img->getDataBuffer(), camera);
    ALICE_TRACER::checkCudaErrors(hipGetLastError());
    ALICE_TRACER::checkCudaErrors(hipDeviceSynchronize());
    stop = clock();

    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "the current frame took " << timer_seconds << " seconds.\n";

    // render to the screen
    while(window.updateWindow()){
        // render to the screen
        texture.update(res_img);
        texture.drawTexture();
        widgets.updateImGui();
        window.swapBuffer();
    }
    widgets.destroyImGui();
    window.releaseWindow();

    res_img->saveImage("../showcases/test.png");
    delete res_img;
    return 0;
}
